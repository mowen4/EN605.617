#include "hip/hip_runtime.h"
/**
* Assignment 4.
* @author: Michael Owen
* Code that will perform simple CUDA operations on data and
* will intentionally cause warp branching for academic purposes
 */
 //CUDA imports
#include "hip/hip_runtime.h"
#include ""
//C imports
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <cassert>
#include <iostream>
#include <memory>

#define LIMIT 8000 
__constant__ int A_ARRAY[LIMIT];
__constant__ int B_ARRAY[LIMIT];

//4 kernels using constant memory
__global__ void addKernel(int* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = A_ARRAY[i] + B_ARRAY[i];
}

__global__ void subKernel(int* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = A_ARRAY[i] - B_ARRAY[i];
}

__global__ void multKernel(int* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = A_ARRAY[i] * B_ARRAY[i];
}

__global__ void modKernel(int* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = A_ARRAY[i] % B_ARRAY[i];
}



//4 kernels using shared memory
__global__ void S_addKernel(int* c, int* a, int* b) {

    __shared__ int shared[LIMIT];

    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    int ci = threadIdx.x;

    int* s_a = &shared[0];
    int* s_b = (int*)&shared[LIMIT/4];
    int* s_c = (int*)&shared[LIMIT/2];

    __syncthreads();

    s_a[ci] = a[ti];
    s_b[ci] = b[ti];

    __syncthreads();

    s_c[ci] = s_a[ci] + s_b[ci];

    c[ti] = s_c[ci];
}

__global__ void S_subKernel(int* c, int* a, int* b) {

    __shared__ int shared[LIMIT];

    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    int ci = threadIdx.x;

    int* s_a = &shared[0];
    int* s_b = (int*)&shared[LIMIT / 4];
    int* s_c = (int*)&shared[LIMIT / 2];

    __syncthreads();

    s_a[ci] = a[ti];
    s_b[ci] = b[ti];

    __syncthreads();

    s_c[ci] = s_a[ci] - s_b[ci];

    c[ti] = s_c[ci];
}

__global__ void S_multKernel(int* c, int* a, int* b) {

    __shared__ int shared[LIMIT];

    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    int ci = threadIdx.x;

    int* s_a = &shared[0];
    int* s_b = (int*)&shared[LIMIT / 4];
    int* s_c = (int*)&shared[LIMIT / 2];

    __syncthreads();

    s_a[ci] = a[ti];
    s_b[ci] = b[ti];

    __syncthreads();

    s_c[ci] = s_a[ci] * s_b[ci];

    c[ti] = s_c[ci];
}

__global__ void S_modKernel(int* c, int* a, int* b) {

    __shared__ int shared[LIMIT];

    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    int ci = threadIdx.x;

    int* s_a = &shared[0];
    int* s_b = (int*)&shared[LIMIT / 4];
    int* s_c = (int*)&shared[LIMIT / 2];

    __syncthreads();

    s_a[ci] = a[ti];
    s_b[ci] = b[ti];

    __syncthreads();

    s_c[ci] = s_a[ci] % s_b[ci];

    c[ti] = s_c[ci];
}

// Helper function for using CUDA
void helperCudaConstant(int* c, const int* a, const int* b, int size, int blocks, int threads) {

    //initialize device memory variables
    int* dev_c = nullptr;

    //Copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(A_ARRAY), a, size * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(B_ARRAY), b, size * sizeof(int));

    //for (int i = 0; i < 10; i++) {
    //    printf("A: %d\tB: %d\n", A_ARRAY[i], B_ARRAY[i]);
    //}

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.

    // Launch Kernels
    addKernel << < blocks, threads >> > (dev_c);

    subKernel << <blocks, threads >> > (dev_c);

    multKernel << < blocks, threads >> > (dev_c);

    modKernel << < blocks, threads >> > (dev_c);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(dev_c);
}

// Helper function for using CUDA
void helperCudaShared(int* c, const int* a, const int* b, int size, int blocks, int threads) {

    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch Kernels
    S_addKernel << < blocks, threads >> > (dev_c, dev_a, dev_b);

    S_subKernel << <blocks, threads >> > (dev_c, dev_a, dev_b);

    S_multKernel << < blocks, threads >> > (dev_c, dev_a, dev_b);

    S_modKernel << < blocks, threads >> > (dev_c, dev_a, dev_b);

    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

//main and driver code

int main(int argc, char** argv) {
    const int arraySize = LIMIT;
    //const unsigned int bytes = arraySize * sizeof(float);
    int blocks = 512;
    int threads = 256;
    int a[arraySize], b[arraySize], c[arraySize];
    clock_t start, end;
    double time_spent;

    //allow for changing number of blocks 
    if (argc == 2) {

        blocks = atoi(argv[1]);
        printf("Blocks changed to:%i\n", blocks);

    }

    //allow for changing number of threads
    else if (argc == 3) {

        blocks = atoi(argv[1]);
        threads = atoi(argv[2]);

        printf("Blocks changed to:%i\n", blocks);
        printf("Threads changed to:%i\n", threads);
    }

    //populate arrays and run cuda code
    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
        b[i] = rand() % 4;
    }

    start = clock();
    helperCudaConstant(c, a, b, arraySize, blocks, threads);
    end = clock();
    time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("\n\nAll constant memory Cuda math operations: %f seconds\n", time_spent);

    hipDeviceReset();

    //populate arrays and run cuda code
    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
        b[i] = rand() % 4;
    }

    start = clock();
    helperCudaShared(c, a, b, arraySize, blocks, threads);
    end = clock();
    time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("All shared memory Cuda math operations: %f seconds\n", time_spent);

    return 0;
}
