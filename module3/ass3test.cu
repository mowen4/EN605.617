#include "hip/hip_runtime.h"
/**
* Assignment 3. 
* @author: Michael Owen
* Code that will perform simple CUDA operations on data and
* will intentionally cause warp branching for academic purposes
 */
//CUDA imports
#include "hip/hip_runtime.h"
#include ""
//C imports
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

__global__ void addKernel(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void subKernel(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
        c[i] = a[i] - b[i];
}

__global__ void multKernel(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
        c[i] = a[i] * b[i];
}

__global__ void modKernel(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
        c[i] = a[i] % b[i];
}

__global__ void addKernelBranch(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x <16) {
        c[i] = a[i] + b[i];
    }
    else {
        c[i] = a[i] + b[i] / threadIdx.x;

    } 
}

__global__ void subKernelBranch(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x < 16) {
        c[i] = a[i] - b[i];
    }
    else {
        c[i] = a[i] - b[i];
        c[i] *= threadIdx.x;
    }
}

__global__ void multKernelBranch(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x < 16) {
        c[i] = a[i] * b[i];
    }
    else {
        c[i] = a[i] * b[i] + threadIdx.x;
    }
}

__global__ void modKernelBranch(int* c, const int* a, const int* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x < 16) {
        c[i] = a[i] % b[i];
    }
    else {
        c[i] = a[i] % b[i] * threadIdx.x;
    }
}

// Helper function for intentional branch warping
void branchingCuda(int* c, const int* a, const int* b, int size, int blocks, int threads) {

    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    // Allocate GPU buffers for three arrays
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy inputs from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    
    //Launch Kernels
    addKernelBranch << < blocks, threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);


    subKernelBranch << < blocks, threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    multKernelBranch << < blocks, threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    modKernelBranch << < blocks, threads >> > (dev_c, dev_a, dev_b);

    hipDeviceSynchronize();

    // Copy output from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}


// Helper function for using CUDA
void helperCuda(int* c, const int* a, const int* b, int size, int blocks, int threads) {
    
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch Kernels
    addKernel << < blocks, threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);


    subKernel <<<blocks, threads >>> (dev_c, dev_a, dev_b);

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    multKernel <<< blocks, threads >>> (dev_c, dev_a, dev_b);

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    modKernel <<< blocks, threads >>> (dev_c, dev_a, dev_b);

    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}


//main driver
int main(int argc, char** argv) {
    const int arraySize = 32000;
    int blocks = 512;
    int threads = 256;
    int a[arraySize], b[arraySize], c[arraySize];

    //allow for changing number of blocks 
    if (argc == 2) {

        blocks = atoi(argv[1]);
        printf("Blocks changed to:%i\n", blocks);

    }

    //allow for changing number of threads
    else if (argc == 3) {

        blocks = atoi(argv[1]);
        threads = atoi(argv[2]);

        printf("Blocks changed to:%i\n", blocks);
        printf("Threads changed to:%i\n", threads);
    }

    //populate arrays and run branching code
    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
        b[i] = rand() % 4;
    }

    printf("Executing with %d blocks and %d threads on array length %d \n", blocks, threads, arraySize);

    clock_t start = clock();
    branchingCuda(c, a, b, arraySize, blocks, threads);
    clock_t end = clock();
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("All branching Cuda operations: %f seconds \n", time_spent);

    hipDeviceReset();
    //populate arrays and run non branching code
    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
        b[i] = rand() % 4;
    }

    clock_t start_branch = clock();
    helperCuda(c, a, b, arraySize, blocks, threads);
    clock_t end_branch = clock();
    double time_spent_branch = (double)(end_branch - start_branch) / CLOCKS_PER_SEC;
    printf("All standard Cuda operations: %f seconds\n", time_spent_branch);

    hipDeviceReset();

    return 0;
}
