#include "hip/hip_runtime.h"
/**
* Assignment 7.
* @author: Michael Owen
* Code that will perform simple CUDA operations on data utilizing the
* registers on the device and streams for async behavior
*
 */
 //CUDA imports
#include "hip/hip_runtime.h"
#include ""
//C imports
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>

//4 kernels using register memory
__global__ void registerAddKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a + reg_b;

    __syncthreads();

    c[ti] = reg_c;
}

__global__ void registerSubKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a - reg_b;

    __syncthreads();

    c[ti] = reg_c;
}

__global__ void registerMultKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a * reg_b;

    __syncthreads();

    c[ti] = reg_c;
}

__global__ void registerModKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a % reg_b;

    __syncthreads();

    c[ti] = reg_c;
}


// Helper function for using CUDA
void asyncCudaRegister(const int* a, const int* b, unsigned int size, int blocks, int threads) {

    //initialize cuda stream objects
    hipStream_t stream1, stream2, stream3, stream4;
    hipEvent_t mem1, mem2, mem3, mem4;
    
    hipEventCreate(&mem1);
    hipEventCreate(&mem2);
    hipEventCreate(&mem3);
    hipEventCreate(&mem4);

    //initialize timing metrics
    clock_t start, end;
    double time_spent;
    start = clock();

    //memory pointers
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c1, * dev_c2, * dev_c3, * dev_c4;
    int* c1, * c2, * c3, * c4;

    //allocate host memory to write back to
    c1 = (int*)malloc(size * sizeof(int));
    c2 = (int*)malloc(size * sizeof(int));
    c3 = (int*)malloc(size * sizeof(int));
    c4 = (int*)malloc(size * sizeof(int));

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c1, size * sizeof(int));
    hipMalloc((void**)&dev_c2, size * sizeof(int));
    hipMalloc((void**)&dev_c3, size * sizeof(int));
    hipMalloc((void**)&dev_c4, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    //create cuda streams, one for each meth kernel
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    // async operations no blocking until after all 
    // copy all data into each stream and recrod events
    hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipEventRecord(mem1, stream1);
    
    hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream2);
    hipEventRecord(mem2, stream2);

    hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream3);
    hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream3);
    hipEventRecord(mem3, stream3);

    hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream4);
    hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream4);
    hipEventRecord(mem4, stream4);

    // launch kernels
    hipStreamWaitEvent(stream1, mem1, 0);
    registerAddKernel << < blocks, threads, 0, stream1 >> > (dev_c1, dev_a, dev_b);
    hipEventRecord(mem1, stream1);

    hipStreamWaitEvent(stream2, mem2, 0);
    registerSubKernel << < blocks, threads, 0, stream2 >> > (dev_c2, dev_a, dev_b);
    hipEventRecord(mem2, stream2);
    
    hipStreamWaitEvent(stream3, mem3, 0);
    registerMultKernel << < blocks, threads, 0, stream3 >> > (dev_c3, dev_a, dev_b);
    hipEventRecord(mem3, stream3);
    
    registerMultKernel << < blocks, threads, 0, stream4 >> > (dev_c4, dev_a, dev_b);
    hipStreamWaitEvent(stream4, mem4, 0);
    hipEventRecord(mem4, stream4);

    hipStreamWaitEvent(stream1, mem1, 0);
    hipStreamWaitEvent(stream2, mem2, 0);
    hipStreamWaitEvent(stream3, mem3, 0);
    hipStreamWaitEvent(stream4, mem4, 0);

    //copy back
    hipMemcpyAsync(c1, dev_c1, size * sizeof(int), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(c2, dev_c2, size * sizeof(int), hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(c3, dev_c3, size * sizeof(int), hipMemcpyDeviceToHost, stream3);
    hipMemcpyAsync(c4, dev_c4, size * sizeof(int), hipMemcpyDeviceToHost, stream4);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    hipStreamSynchronize(stream4);

    for (int i = 0; i < 10; i++) {
        printf("%d\t%d\t%d\t%d\t%d\t%d\n", a[i], b[i], c1[i], c2[i], c3[i], c4[i]);
    }

    for (int i = size - 10; i < size; i++) {
        printf("%d\t%d\t%d\t%d\t%d\t%d\n", a[i], b[i], c1[i], c2[i], c3[i], c4[i]);
    }

    end = clock();
    time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("\n\nData size %i register memory Cuda math operations: %f seconds\n", size, time_spent);

    //free memory
    free(c1);
    free(c2);
    free(c3);
    free(c4);
    hipFree(dev_a);
    hipFree(dev_b);

    //close streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
}


// function for populating a and b arrays with specified data
__host__ void generateData(int* a, int* b, int arraySize) {

    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
        b[i] = rand() % 4;
    }
}

//main and driver code
int main(int argc, char** argv) {
    unsigned int arraySize = 10000;
    int blocks = 400;
    int threads = 256;
    int* h_a, * h_b;
    //allow for changing number of threads
    if (argc == 4) {

        arraySize = atoi(argv[1]);
        blocks = atoi(argv[2]);
        threads = atoi(argv[3]);

        printf("Array Length changed to:%i\n", arraySize);
        printf("Blocks changed to:%i\n", blocks);
        printf("Threads changed to:%i\n", threads);

    }

    //allocate pinned memory for copying to device
    hipHostMalloc((void**)&h_a, arraySize * sizeof(int));
    hipHostMalloc((void**)&h_b, arraySize * sizeof(int));

    //poppulate memory with random values
    generateData(h_a, h_b, arraySize);

    //async math kernel calls
    asyncCudaRegister(h_a, h_b, arraySize, blocks, threads);

    //free pinned memory
    hipHostFree(h_a);
    hipHostFree(h_b);

    return 0;
}
