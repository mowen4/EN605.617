#include "hip/hip_runtime.h"
/**
* Assignment 7.
* @author: Michael Owen
* Code that will perform simple CUDA operations on data utilizing the
* registers on the device and streams for async behavior
*
 */
 //CUDA imports
#include "hip/hip_runtime.h"
#include ""
//C imports
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>

//4 kernels using register memory
__global__ void registerAddKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a + reg_b;

    __syncthreads();

    c[ti] = reg_c;
}

__global__ void registerSubKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a - reg_b;

    __syncthreads();

    c[ti] = reg_c;
}

__global__ void registerMultKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a * reg_b;

    __syncthreads();

    c[ti] = reg_c;
}

__global__ void registerModKernel(int* c, int* a, int* b) {

    int ti = blockIdx.x * blockDim.x + threadIdx.x;

    int reg_a = a[ti];
    int reg_b = b[ti];
    int reg_c = 0;

    __syncthreads();

    reg_c = reg_a % reg_b;

    __syncthreads();

    c[ti] = reg_c;
}


// Helper function for using CUDA
void asyncCudaRegister(const int* a, const int* b, unsigned int size, int blocks, int threads) {

    //initialize cuda stream objects
    hipStream_t stream1, stream2, stream3, stream4;
    hipEvent_t mem1, mem2, mem3, mem4;
    
    hipEventCreate(&mem1);
    hipEventCreate(&mem2);
    hipEventCreate(&mem3);
    hipEventCreate(&mem4);

    //initialize timing metrics
    clock_t start, end;
    double time_spent;
    start = clock();

    //memory pointers
    int* dev_a1, * dev_b1, * dev_a2, * dev_b2, * dev_a3, * dev_b3, * dev_a4, * dev_b4;
    int* dev_c1, * dev_c2, * dev_c3, * dev_c4;
    int* c1, *c2, *c3, *c4;

    //allocate host memory to write back to
    c1 = (int*)malloc(size * sizeof(int));
    c2 = (int*)malloc(size * sizeof(int));
    c3 = (int*)malloc(size * sizeof(int));
    c4 = (int*)malloc(size * sizeof(int));

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c1, size * sizeof(int));
    hipMalloc((void**)&dev_c2, size * sizeof(int));
    hipMalloc((void**)&dev_c3, size * sizeof(int));
    hipMalloc((void**)&dev_c4, size * sizeof(int));
    hipMalloc((void**)&dev_a1, size * sizeof(int));
    hipMalloc((void**)&dev_b1, size * sizeof(int));
    hipMalloc((void**)&dev_a2, size * sizeof(int));
    hipMalloc((void**)&dev_b2, size * sizeof(int));
    hipMalloc((void**)&dev_a3, size * sizeof(int));
    hipMalloc((void**)&dev_b3, size * sizeof(int));
    hipMalloc((void**)&dev_a4, size * sizeof(int));
    hipMalloc((void**)&dev_b4, size * sizeof(int));

    //create cuda streams, one for each meth kernel
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    // async operations no blocking until after all 
    // copy all data into each stream and recrod events
    hipMemcpyAsync(dev_a1, a, size * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_b1, b, size * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipEventRecord(mem1, stream1);
    
    hipMemcpyAsync(dev_a2, a, size * sizeof(int), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(dev_b2, b, size * sizeof(int), hipMemcpyHostToDevice, stream2);
    hipEventRecord(mem2, stream2);

    hipMemcpyAsync(dev_a3, a, size * sizeof(int), hipMemcpyHostToDevice, stream3);
    hipMemcpyAsync(dev_b3, b, size * sizeof(int), hipMemcpyHostToDevice, stream3);
    hipEventRecord(mem3, stream3);

    hipMemcpyAsync(dev_a4, a, size * sizeof(int), hipMemcpyHostToDevice, stream4);
    hipMemcpyAsync(dev_b4, b, size * sizeof(int), hipMemcpyHostToDevice, stream4);
    hipEventRecord(mem4, stream4);

    // launch kernels
    hipDeviceSynchronize();

    hipStreamWaitEvent(stream1, mem1);
    registerAddKernel << < blocks, threads, 0, stream1 >> > (dev_c1, dev_a1, dev_b1);
    hipEventRecord(mem1, stream1);

    hipStreamWaitEvent(stream2, mem2);
    registerSubKernel << < blocks, threads, 0, stream2 >> > (dev_c2, dev_a2, dev_b2);
    hipEventRecord(mem2, stream2);
    
    hipStreamWaitEvent(stream3, mem3);
    registerMultKernel << < blocks, threads, 0, stream3 >> > (dev_c3, dev_a3, dev_b3);
    hipEventRecord(mem3, stream3);
    
    hipStreamWaitEvent(stream4, mem4);
    registerModKernel << < blocks, threads, 0, stream4 >> > (dev_c4, dev_a4, dev_b4);
    hipEventRecord(mem4, stream4);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    hipStreamSynchronize(stream4);

    hipDeviceSynchronize();

    //hipStreamWaitEvent(stream1, mem1, 0);
    //hipStreamWaitEvent(stream2, mem2, 0);
    //hipStreamWaitEvent(stream3, mem3, 0);
    //hipStreamWaitEvent(stream4, mem4, 0);

    //copy back
    hipMemcpy(c1, dev_c1, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c2, dev_c2, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c3, dev_c3, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c4, dev_c4, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d\t%d\t%d\t%d\t%d\t%d\n", a[i], b[i], c1[i], c2[i], c3[i], c4[i]);
    }

    for (int i = size - 10; i < size; i++) {
        printf("%d\t%d\t%d\t%d\t%d\t%d\n", a[i], b[i], c1[i], c2[i], c3[i], c4[i]);
    }

    end = clock();
    time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("\n\nData size %i register memory Cuda math operations: %f seconds\n", size, time_spent);

    //free memory
    free(c1);
    free(c2);
    free(c3);
    free(c4);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_a2);
    hipFree(dev_b2);
    hipFree(dev_a3);
    hipFree(dev_b3);
    hipFree(dev_a4);
    hipFree(dev_b4);
    hipFree(dev_c1);
    hipFree(dev_c2);
    hipFree(dev_c3);
    hipFree(dev_c4);


    //close streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
}


// function for populating a and b arrays with specified data
__host__ void generateData(int* a, int* b, int arraySize) {

    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
        b[i] = rand() % 4;
    }
}

//main and driver code
int main(int argc, char** argv) {
    unsigned int arraySize = 10000;
    int blocks = 400;
    int threads = 256;
    int* h_a, * h_b;
    //allow for changing number of threads
    if (argc == 4) {

        arraySize = atoi(argv[1]);
        blocks = atoi(argv[2]);
        threads = atoi(argv[3]);

        printf("Array Length changed to:%i\n", arraySize);
        printf("Blocks changed to:%i\n", blocks);
        printf("Threads changed to:%i\n", threads);

    }

    //allocate pinned memory for copying to device
    hipHostMalloc((void**)&h_a, arraySize * sizeof(int));
    hipHostMalloc((void**)&h_b, arraySize * sizeof(int));

    //poppulate memory with random values
    generateData(h_a, h_b, arraySize);

    //async math kernel calls
    asyncCudaRegister(h_a, h_b, arraySize, blocks, threads);

    //free pinned memory
    hipHostFree(h_a);
    hipHostFree(h_b);

    return 0;
}
